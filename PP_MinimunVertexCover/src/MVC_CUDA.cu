#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <string>
#include <vector>
#include "Grafo.h"
#include "MVC_Serial.h"

using namespace std;

#define MAX_THREADS_BY_BLOCK 1024

bool *mvc;
int nEjecCUD, nEjecZero;

struct structFile{
	string nameFile;
	bool isCSV;
	int iniUno;
	structFile(string a, bool b, int c){ nameFile = a; isCSV = b; iniUno = c;}
};


__device__ int getIdVertex() {
	return threadIdx.x + blockIdx.x * blockDim.x;
}

//__global__ void kernel1_mvc(int nNodes, listNode *nodes, int *listNeigh, bool *mvc) {
__global__ void kernel1_mvc(int nNodes, nodeSimple *nodes, int *listNeigh, bool *mvc) {
	int tid = getIdVertex();
	if(tid < nNodes){
		int deg = nodes[tid].grado;
		int posVec = nodes[tid].posIniNei;
		int tempDeg, mdeg = deg;

		for(int i = 0; i < deg; i++){
			tempDeg = nodes[listNeigh[posVec + i]].grado;
			mdeg = min(mdeg, tempDeg);
		}
		if(deg == mdeg)
			mvc[tid] = false;
	}
}

//__global__ void kernel2_mvc(int nNodes, listNode *nodes, int *listNeigh, bool *mvc, bool *adj, bool *prevMvc, bool *terminated) {
__global__ void kernel2_mvc(int nNodes, nodeSimple *nodes, int *listNeigh, bool *mvc, bool *adj, bool *prevMvc, bool *terminated) {
	int tid = getIdVertex();
	if(tid < nNodes){
		int nEdges = nodes[tid].grado;
		int posVec = nodes[tid].posIniNei;

		adj[tid] = true;
		for(int i = 0; i < nEdges; i++)
			if(!mvc[listNeigh[posVec + i]])
				adj[tid] = false;
		if(mvc[tid] != !adj[tid])
			*terminated = false;
		prevMvc[tid] = mvc[tid];
	}
}

//__global__ void kernel3_mvc(int nNodes, listNode *nodes, int *listNeigh, bool *mvc, bool *adj, bool *prevMvc) {
__global__ void kernel3_mvc(int nNodes, nodeSimple *nodes, int *listNeigh, bool *mvc, bool *adj, bool *prevMvc) {
	int tid = getIdVertex(), eid;
	if(tid < nNodes){
		int nEdges = nodes[tid].grado;
		int posVec = nodes[tid].posIniNei;

		for(int i = 0; i < nEdges; i++) {
			eid = listNeigh[posVec + i];
			if(prevMvc[eid] && !adj[eid] && adj[tid])
				mvc[tid] = false;
		}
	}
}

//__global__ void kernel4_mvc(int nNodes, listNode *nodes, int *listNeigh, bool *mvc, bool *adj, bool *prevMvc) {
__global__ void kernel4_mvc(int nNodes, nodeSimple *nodes, int *listNeigh, bool *mvc, bool *adj, bool *prevMvc) {
	int tid = getIdVertex();
	if(tid < nNodes){
		if(!prevMvc[tid] && !adj[tid]){
			int nEdges = nodes[tid].grado, eid;
			int posVec = nodes[tid].posIniNei;

			for(int i = 0; i < nEdges; i++){
				eid = listNeigh[posVec + i];
				if(!prevMvc[eid] && !adj[eid] && eid < tid)
					mvc[tid] = true;
			}
		}
	}
}

float ejecutarCUDAZeroCopy(Graph *grafo) {
	// variables para host
	bool *adj, *prevMvc, *terminated;
	//listNode *nodes;
	nodeSimple *nodes;
	int *listNeigh;
	int nNodes = grafo->numVert;
	int nEdges = grafo->numEdges;

	nEjecZero = 0;

	// variables para device
	//listNode *devNodes;
	nodeSimple *devNodes;
	bool *devMvc, *devPrevMvc, *devAdj, *devTerminated;
	int *devListNeig;

	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	hipHostAlloc((void**)&terminated, sizeof(bool), hipHostMallocMapped);
	hipHostAlloc((void**)&mvc, nNodes * sizeof(bool), hipHostMallocMapped);
	hipHostAlloc((void**)&adj, nNodes * sizeof(bool), hipHostMallocMapped);
	hipHostAlloc((void**)&prevMvc, nNodes * sizeof(bool), hipHostMallocWriteCombined | hipHostMallocMapped);
	//hipHostAlloc((void**)&nodes, nNodes * sizeof(listNode), hipHostMallocWriteCombined | hipHostMallocMapped);
	hipHostAlloc((void**)&nodes, nNodes * sizeof(nodeSimple), hipHostMallocWriteCombined | hipHostMallocMapped);
	hipHostAlloc((void**)&listNeigh, nEdges * sizeof(int), hipHostMallocWriteCombined | hipHostMallocMapped);

	hipHostGetDevicePointer(&devTerminated, terminated, 0);
	hipHostGetDevicePointer(&devMvc, mvc, 0);
	hipHostGetDevicePointer(&devAdj, adj, 0);
	hipHostGetDevicePointer(&devPrevMvc, prevMvc, 0);
	hipHostGetDevicePointer(&devNodes, nodes, 0);
	hipHostGetDevicePointer(&devListNeig, listNeigh, 0);

	// Se asignan los valores iniciales de cada variable
	*terminated = false;
	for(uint i = 0; i < nNodes; i++) {
		//nodes[i] = grafo->vert[i];
		nodes[i] = grafo->listNodeSimple[i];
		mvc[i] = true;
		adj[i] = true;
		prevMvc[i] = true;
	}
	for(uint i = 0; i < nEdges; i++) {
		listNeigh[i] = grafo->listNeight[i];
	}

	int blocks = (nNodes + MAX_THREADS_BY_BLOCK - 1) / MAX_THREADS_BY_BLOCK;
	int threads = MAX_THREADS_BY_BLOCK;

	kernel1_mvc<<<blocks, threads>>>(nNodes, devNodes, devListNeig, devMvc);
	hipDeviceSynchronize();
	kernel2_mvc<<<blocks, threads>>>(nNodes, devNodes, devListNeig, devMvc, devAdj, devPrevMvc, devTerminated);
	hipDeviceSynchronize();
	while(!(*terminated)) {
		nEjecZero++;
		*terminated = true;
		kernel3_mvc<<<blocks, threads>>>(nNodes, devNodes, devListNeig, devMvc, devAdj, devPrevMvc);
		hipDeviceSynchronize();
		kernel2_mvc<<<blocks, threads>>>(nNodes, devNodes, devListNeig, devMvc, devAdj, devPrevMvc, devTerminated);
		hipDeviceSynchronize();
		kernel4_mvc<<<blocks, threads>>>(nNodes, devNodes, devListNeig, devMvc, devAdj, devPrevMvc);
		hipDeviceSynchronize();
		kernel2_mvc<<<blocks, threads>>>(nNodes, devNodes, devListNeig, devMvc, devAdj, devPrevMvc, devTerminated);
		hipDeviceSynchronize();
	}

	hipHostFree(mvc);
	hipHostFree(adj);
	hipHostFree(prevMvc);
	hipHostFree(listNeigh);
	hipHostFree(nodes);
	hipHostFree(&terminated);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return elapsedTime / 1000.0;
}

float ejecutarCUDA(Graph* grafo) {
	// variables para host
	bool *adj, terminated = false;
	int nNodes = grafo->numVert;
	int nEdges = grafo->numEdges;

	nEjecCUD = 0;

	// variables para device
	//listNode *devNodes;
	nodeSimple *devNodes;
	bool *devMvc, *devPrevMvc, *devAdj, *devTerminated;
	int *devListNeig;

	mvc = new bool[nNodes];
	adj = new bool[nNodes];

	for(uint i = 0; i < nNodes; i++) {
		mvc[i] = true;
		adj[i] = true;
	}

	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	hipMalloc((void**)&devTerminated, sizeof(bool));
	hipMalloc((void**)&devMvc, nNodes * sizeof(bool));
	hipMalloc((void**)&devPrevMvc, nNodes * sizeof(bool));
	hipMalloc((void**)&devAdj, nNodes * sizeof(bool));
	//hipMalloc((void**)&devNodes, nNodes * sizeof(listNode));
	hipMalloc((void**)&devNodes, nNodes * sizeof(nodeSimple));
	hipMalloc((void**)&devListNeig, nEdges * sizeof(int));

	hipMemcpy(devTerminated, &terminated, sizeof(bool), hipMemcpyHostToDevice);
	hipMemcpy(devMvc, mvc, nNodes * sizeof(bool), hipMemcpyHostToDevice);
	hipMemcpy(devPrevMvc, mvc, nNodes * sizeof(bool), hipMemcpyHostToDevice);
	hipMemcpy(devAdj, adj, nNodes * sizeof(bool), hipMemcpyHostToDevice);
	//hipMemcpy(devNodes, grafo->vert, nNodes * sizeof(listNode), hipMemcpyHostToDevice);
	hipMemcpy(devNodes, grafo->listNodeSimple, nNodes * sizeof(nodeSimple), hipMemcpyHostToDevice);
	hipMemcpy(devListNeig, grafo->listNeight, nEdges * sizeof(int), hipMemcpyHostToDevice);

	int blocks = (nNodes + MAX_THREADS_BY_BLOCK - 1) / MAX_THREADS_BY_BLOCK;
	int threads = MAX_THREADS_BY_BLOCK;

	kernel1_mvc<<<blocks, threads>>>(nNodes, devNodes, devListNeig, devMvc);
	kernel2_mvc<<<blocks, threads>>>(nNodes, devNodes, devListNeig, devMvc, devAdj, devPrevMvc, devTerminated);
	while(!terminated){
		nEjecCUD++;
		terminated = true;
		hipMemcpy(devTerminated, &terminated, sizeof(bool), hipMemcpyHostToDevice);
		kernel3_mvc<<<blocks, threads>>>(nNodes, devNodes, devListNeig, devMvc, devAdj, devPrevMvc);
		kernel2_mvc<<<blocks, threads>>>(nNodes, devNodes, devListNeig, devMvc, devAdj, devPrevMvc, devTerminated);
		kernel4_mvc<<<blocks, threads>>>(nNodes, devNodes, devListNeig, devMvc, devAdj, devPrevMvc);
		kernel2_mvc<<<blocks, threads>>>(nNodes, devNodes, devListNeig, devMvc, devAdj, devPrevMvc, devTerminated);
		hipMemcpy(&terminated, devTerminated, sizeof(bool), hipMemcpyDeviceToHost);
	}
	hipMemcpy(mvc, devMvc, nNodes * sizeof(bool), hipMemcpyDeviceToHost);
	hipMemcpy(adj, devAdj, nNodes * sizeof(bool), hipMemcpyDeviceToHost);

	hipFree(devMvc);
	hipFree(devPrevMvc);
	hipFree(devAdj);
	hipFree(devListNeig);
	hipFree(devNodes);
	hipFree(devTerminated);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return elapsedTime / 1000.0;
}

int main() {
	string path = "data/";
	string resPath = "res/";
	vector<structFile> arrayFiles;
//	arrayFiles.push_back(structFile("randomGraph4.csv", 1, 1));
//	arrayFiles.push_back(structFile("randomGraph6_01.csv", 1, 1));
//	arrayFiles.push_back(structFile("randomGraph7_01.csv", 1, 1));
//	arrayFiles.push_back(structFile("randomGraph7_02.csv", 1, 1));
//	arrayFiles.push_back(structFile("randomGraph10.csv", 1, 1));
	arrayFiles.push_back(structFile("randomGraph10000.csv", 1, 1));
	arrayFiles.push_back(structFile("p2p-Gnutella31.txt", 0, 0));
	arrayFiles.push_back(structFile("networkGraph_20000_600000.csv", 1, 1));
//	arrayFiles.push_back(structFile("web-BerkStan_685230_7600595.txt", 0, 1));
//	arrayFiles.push_back(structFile("soc-LiveJournal1.txt", 0, 0));

	int whichDevice;
	hipDeviceProp_t prop;
	hipGetDevice(&whichDevice);
	hipGetDeviceProperties(&prop, whichDevice);
	if(prop.canMapHostMemory != 1) {
		cout << "Device no puede mapear memoria en CPU" << endl;
		return 0;
	}
	hipSetDeviceFlags(hipDeviceMapHost);

	float elapsedTime1, elapsedTime2;
	for(int i = 0; i < arrayFiles.size(); i++) {
		Graph* g = new Graph();
		g->levantarGrafo((path + arrayFiles[i].nameFile).c_str(), arrayFiles[i].isCSV, arrayFiles[i].iniUno);
		g->refinarGrafo();
		g->compactarGrafo();

		// Ejecutando version CUDA con global memory
		elapsedTime1 = ejecutarCUDA(g);
		g->genFileForVisualization((resPath + arrayFiles[i].nameFile + ".graphml").c_str(), mvc);

		// Ejecutando version CUDA con zero-memory
		elapsedTime2 = ejecutarCUDAZeroCopy(g);

		// Ejecutando version SERIAL
		MVCSerial mvcSerial(*g);
		mvcSerial.ejecutarSerial();
		bool *arrayMVCSerial = mvcSerial.getListNodesMVC();
		int nNodesMVCSerial = mvcSerial.getnNodesMVC();

		printf("Graph: %s - nVertex: %d\n", arrayFiles[i].nameFile.c_str(), g->numVert);
		printf("> Times: CUDA GloMem = %f secs, CUDA ZeroMem = %f secs, Serial = %f secs. \n", elapsedTime1, elapsedTime2, mvcSerial.getTimeExe());
		printf("%d %d %d\n", nEjecCUD, nEjecZero, mvcSerial.nEjec);
	}
	return 0;
}
