#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <string>
#include "Grafo.h"
#include "MVC_Serial.h"

using namespace std;

#define MAX_THREADS_BY_BLOCK 1024

int nNodesMCVCUDA, *arrayMvcCUDA;
float elapsedTime;

__global__ void knowData(long *x1, long *y1, long *z1, long *x2, long *y2, long *z2){
	*x1 = (long)blockDim.x;
	*y1 = (long)blockDim.y;
	*z1 = (long)blockDim.z;

	*x2 = (long)gridDim.x;
	*y2 = (long)gridDim.y;
	*z2 = (long)gridDim.z;

}

__device__ int getIdVertex() {
	return threadIdx.x + blockIdx.x * blockDim.x;
}

__global__ void kernel1_mvc(int* nNodes, listNode *nodes, int *listNeigh, bool *mvc) {
	int tid = getIdVertex();
	if(tid < *nNodes){
		int deg = nodes[tid].grado;
		int posVec = nodes[tid].posIniNei;
		int tempDeg, mdeg = deg;

		for(int i = 0; i < deg; i++){
			tempDeg = nodes[listNeigh[posVec + i]].grado;
			mdeg = min(mdeg, tempDeg);
		}
		if(deg == mdeg)
			mvc[tid] = false;
	}
}

__global__ void kernel2_mvc(int* nNodes, listNode *nodes, int *listNeigh, bool *mvc, bool *adj, bool *prevMvc, bool *terminated) {
	int tid = getIdVertex();
	if(tid < *nNodes){
		int nEdges = nodes[tid].grado;
		int posVec = nodes[tid].posIniNei;

		adj[tid] = true;
		for(int i = 0; i < nEdges; i++)
			if(!mvc[listNeigh[posVec + i]])
				adj[tid] = false;
		if(mvc[tid] != !adj[tid])
			*terminated = false;
		prevMvc[tid] = mvc[tid];
	}
}

__global__ void kernel3_mvc(int *nNodes, listNode *nodes, int *listNeigh, bool *mvc, bool *adj, bool *prevMvc) {
	int tid = getIdVertex(), eid;
	if(tid < *nNodes){
		int nEdges = nodes[tid].grado;
		int posVec = nodes[tid].posIniNei;

		for(int i = 0; i < nEdges; i++) {
			eid = listNeigh[posVec + i];
			if(prevMvc[eid] && !adj[eid] && adj[tid])
				mvc[tid] = false;
		}
	}
}

__global__ void kernel4_mvc(int *nNodes, listNode *nodes, int *listNeigh, bool *mvc, bool *adj, bool *prevMvc) {
	int tid = getIdVertex();
	if(tid < *nNodes){
		if(!prevMvc[tid] && !adj[tid]){
			int nEdges = nodes[tid].grado, eid;
			int posVec = nodes[tid].posIniNei;

			for(int i = 0; i < nEdges; i++){
				eid = listNeigh[posVec + i];
				if(!prevMvc[eid] && !adj[eid] && eid < tid)
					mvc[tid] = true;
			}
		}
	}
}

void ejecutarCUDA(Graph* grafo) {
	// variables host
	bool *adj, *mvc, terminated = false;
	int nNodes = grafo->numVert;
	int nEdges = grafo->numEdges;

	// variables devices
	listNode *devNodes;
	bool *devMvc, *devPrevMvc, *devAdj, *devTerminated;
	int *devListNeig, *devNumNodes;

	mvc = new bool[nNodes];
	adj = new bool[nNodes];

	for(uint i = 0; i < nNodes; i++) {
		mvc[i] = true;
		adj[i] = true;
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	hipMalloc((void**)&devNumNodes, sizeof(int));
	hipMalloc((void**)&devTerminated, sizeof(bool));
	hipMalloc((void**)&devMvc, nNodes * sizeof(bool));
	hipMalloc((void**)&devPrevMvc, nNodes * sizeof(bool));
	hipMalloc((void**)&devAdj, nNodes * sizeof(bool));
	hipMalloc((void**)&devNodes, nNodes * sizeof(listNode));
	hipMalloc((void**)&devListNeig, nEdges * sizeof(int));

	hipMemcpy(devNumNodes, &nNodes, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(devTerminated, &terminated, sizeof(bool), hipMemcpyHostToDevice);
	hipMemcpy(devMvc, mvc, nNodes * sizeof(bool), hipMemcpyHostToDevice);
	hipMemcpy(devPrevMvc, mvc, nNodes * sizeof(bool), hipMemcpyHostToDevice);
	hipMemcpy(devAdj, adj, nNodes * sizeof(bool), hipMemcpyHostToDevice);
	hipMemcpy(devNodes, grafo->vert, nNodes * sizeof(listNode), hipMemcpyHostToDevice);
	hipMemcpy(devListNeig, grafo->listNeight, nEdges * sizeof(int), hipMemcpyHostToDevice);

	int blocks = (nNodes + MAX_THREADS_BY_BLOCK - 1) / MAX_THREADS_BY_BLOCK;
	int threads = MAX_THREADS_BY_BLOCK;

	kernel1_mvc<<<blocks, threads>>>(devNumNodes, devNodes, devListNeig, devMvc);
	kernel2_mvc<<<blocks, threads>>>(devNumNodes, devNodes, devListNeig, devMvc, devAdj, devPrevMvc, devTerminated);
	while(!terminated){
		terminated = true;
		hipMemcpy(devTerminated, &terminated, sizeof(bool), hipMemcpyHostToDevice);
		kernel3_mvc<<<blocks, threads>>>(devNumNodes, devNodes, devListNeig, devMvc, devAdj, devPrevMvc);
		kernel2_mvc<<<blocks, threads>>>(devNumNodes, devNodes, devListNeig, devMvc, devAdj, devPrevMvc, devTerminated);
		kernel4_mvc<<<blocks, threads>>>(devNumNodes, devNodes, devListNeig, devMvc, devAdj, devPrevMvc);
		kernel2_mvc<<<blocks, threads>>>(devNumNodes, devNodes, devListNeig, devMvc, devAdj, devPrevMvc, devTerminated);
		hipMemcpy(&terminated, devTerminated, sizeof(bool), hipMemcpyDeviceToHost);
	}

	hipMemcpy(mvc, devMvc, nNodes * sizeof(bool), hipMemcpyDeviceToHost);
	hipMemcpy(adj, devAdj, nNodes * sizeof(bool), hipMemcpyDeviceToHost);

	hipFree(devMvc);
	hipFree(devPrevMvc);
	hipFree(devAdj);
	hipFree(devListNeig);
	hipFree(devNodes);
	hipFree(devTerminated);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	elapsedTime = elapsedTime / 1000.0;
	nNodesMCVCUDA = 0;
	for(int i = 0; i < nNodes; i++)
		nNodesMCVCUDA += mvc[i];
	arrayMvcCUDA = new int[nNodesMCVCUDA];
	for(int i = 0, j = 0; i < nNodes; i++)
		if(mvc[i])
			arrayMvcCUDA[j++] = i;
}

int main() {
	string path = "data/";
	string arrayFiles[] = {"randomGraph4.csv", "randomGraph10.csv", "randomGraph7_01.csv", "randomGraph7_02.csv", "randomGraph10.csv", "randomGraph10000.csv"};
	for(int i = 0; i < 6; i++){
		Graph* g = new Graph();
		g->levantarGrafo((path + arrayFiles[i]).c_str());
		g->refinarGrafo();
		g->compactarGrafo();
		ejecutarCUDA(g);

		MVCSerial mvcSerial(*g);
		mvcSerial.ejecutarSerial();
		int *arrayMVCSerial = mvcSerial.getListNodesMVC();
		int nNodesMVCSerial = mvcSerial.getnNodesMVC();
		for(int i = 0; i < nNodesMVCSerial; i++)
			printf("%d%c", arrayMVCSerial[i], i + 1 == nNodesMVCSerial ? '\n' : ' ');

		printf("Graph  nVertices: %d time for CUDA: %f s. Serial: %f s.\n", g->numVert, elapsedTime, mvcSerial.getTimeExe());
	}
	return 0;
}
